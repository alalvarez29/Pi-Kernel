#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <iomanip>
#include <cmath>

const double PI = 3.14159265358979323846;
const long STEP_NUM = 32768 * 32768;
const double STEP_LENGTH = 1.0 / STEP_NUM;
const int THREAD_NUM = 512;
const int BLOCK_NUM = 64;
const int NREPEAT = 50;

__global__ void integrate(float *globalSum, int stepNum, float stepLength, int threadNum, int blockNum)
{
    int globalThreadId = threadIdx.x + blockIdx.x * blockDim.x;
    int start = (stepNum / (blockNum * threadNum)) * globalThreadId;
    int end = (stepNum / (blockNum * threadNum)) * (globalThreadId + 1);
    int localThreadId = threadIdx.x;
    int blockId = blockIdx.x;

    __shared__ float blockSum[THREAD_NUM];

    memset(blockSum, 0, threadNum * sizeof(float));

    double x;
    for(int i = start; i < end; i++)
    {
        x = (i + 0.5) * stepLength;
        blockSum[localThreadId] += 1.0 / (1.0 + x * x);
    }
    blockSum[localThreadId] *= stepLength * 4;

    __syncthreads();

    for(int i = blockDim.x / 2; i > 0; i >>= 1)
    {
        if(localThreadId < i)
        {
            blockSum[localThreadId] += blockSum[localThreadId + i];
        }

        __syncthreads();
    }

    if(localThreadId == 0)
    {
        globalSum[blockId] = blockSum[0];
    }
}

__global__ void sumReduce(double *sum, double *sumArray, int arraySize)
{
    int localThreadId = threadIdx.x;

    for(int i = blockDim.x / 2; i > 0; i >>= 1)
    {
        if (localThreadId < i)
        {
            sumArray[localThreadId] += sumArray[localThreadId + i];
        }

        __syncthreads();
    }

    if(localThreadId == 0)
    {
        *sum = sumArray[0];
    }
}

int main()
{
    int deviceCount = 0;

    std::cout << "Configuring device..." << std::endl;

    hipError_t error = hipGetDeviceCount(&deviceCount);

    if(error != hipSuccess)
    {
        std::cout << "hipGetDeviceCount returned" << (int)error << std::endl;
        std::cout << hipGetErrorString(error) << std::endl;
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error, hipGetErrorString(error));
        return 1;
    }

    if(deviceCount == 0)
    {
        std::cout << "There are no available CUDA devices(s)" << std::endl;
        return 1;
    }
    else
    {
        std::cout << "CUDA Capable device(s) detected " << deviceCount << std::endl;
    }

    double pi = 0.0;
    double *deviceBlockSum;
    double *devicePi;

    //allocate memory on GPU (device)
    hipMalloc((void **) &devicePi, sizeof(double));
    hipMalloc((void **) &deviceBlockSum, sizeof(double) * BLOCK_NUM);

    //timer 
    hipEvent_t startTime, stopTime;
    hipEventCreate(&startTime);
    hipEventCreate(&stopTime);
    hipEventRecord(startTime, 0);
    std::cout << "Approximate pi using a Riemann sum" << std::endl;
    std::cout << std::endl;

    std::cout << "Running CUDA pi approximation" << std::endl;

    for(int repeat = 0; repeat < NREPEAT; repeat++)
    {
        integrate<<<BLOCK_NUM, THREAD_NUM>>>(deviceBlockSum, STEP_NUM, STEP_LENGTH, THREAD_NUM, BLOCK_NUM);
        sumReduce<<<1, BLOCK_NUM>>>(devicePi, deviceBlockSum, BLOCK_NUM);

        if(repeat == (NREPEAT - 1))
        {
            //get result to host from device
            hipMemcpy(&pi, devicePi, sizeof(double), hipMemcpyDeviceToHost);

            std::cout << "\tpi = " << std::setprecision(20) << pi << std::endl;
            std::cout << "\terror = " << std::fixed << fabs(pi - PI) << std::endl;
        }
    }
    hipEventRecord(stopTime, 0);
    hipEventSynchronize(stopTime);
    float gpuTime = 0;
    hipEventElapsedTime(&gpuTime, startTime, stopTime);

	std::cout << "Time elapsed to get the result: " << gpuTime / 1000 << " seconds" << std::endl;
	std::cout << std::endl;

    //free memory
    hipFree(deviceBlockSum);

    hipDeviceReset();
    return 0;
}