#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <assert.h>

const float PI = 3.1415926535897932;
const long STEP_NUM = 32768 * 32768;
const float STEP_LENGTH = 1.0 / STEP_NUM;
const int THREAD_NUM = 512;
const int BLOCK_NUM = 64;

__global__ void integrate(float *globalSum, int stepNum, float stepLength, int threadNum, int blockNum)
{
    int globalThreadId = threadIdx.x + blockIdx.x * blockDim.x;
    int start = (stepNum / (blockNum * threadNum)) * globalThreadId;
    int end = (stepNum / (blockNum * threadNum)) * (globalThreadId + 1);
    int localThreadId = threadIdx.x;
    int blockId = blockIdx.x;

    __shared__ float blockSum[THREAD_NUM];

    memset(blockSum, 0, threadNum * sizeof(float));

    float x;
    for(int i = start; i < end; i++)
    {
        x = (i + 0.5f) * stepLength;
        blockSum[localThreadId] += 1.0f / (1.0f + x * x);
    }
    blockSum[localThreadId] *= stepLength * 4;

    __syncthreads();

    for(int i = blockDim.x / 2; i > 0; i >>= 1)
    {
        if(localThreadId < i)
        {
            blockSum[localThreadId] += blockSum[localThreadId + i];
        }

        __syncthreads();
    }

    if(localThreadId == 0)
    {
        globalSum[blockId] = blockSum[0];
    }
}

__global__ void sumReduce(float *sum, float *sumArray, int arraySize)
{
    int localThreadId = threadIdx.x;

    for(int i = blockDim.x / 2; i > 0; i >>= 1)
    {
        if (localThreadId < i)
        {
            sumArray[localThreadId] += sumArray[localThreadId + i];
        }

        __syncthreads();
    }

    if(localThreadId == 0)
    {
        *sum = sumArray[0];
    }
}

int main()
{
    int deviceCount = 0;

    printf("\nConfiguring device...\n");

    hipError_t error = hipGetDeviceCount(&deviceCount);

    if(error != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error, hipGetErrorString(error));
        return 1;
    }

    if(deviceCount == 0)
    {
        printf("There are no available CUDA device(s)\n");
        return 1;
    }
    else
    {
        printf("%d CUDA Capable device(s) detected\n", deviceCount);
    }

    float pi = 0.0;
    float *deviceBlockSum;
    float *devicePi;

    //allocate memory on GPU (device)
    hipMalloc((void **) &devicePi, sizeof(float));
    hipMalloc((void **) &deviceBlockSum, sizeof(float) * BLOCK_NUM);

    //timer 
    hipEvent_t startTime, stopTime;
    hipEventCreate(&startTime);
    hipEventCreate(&stopTime);
    hipEventRecord(startTime, 0);
    printf("Approximate pi using a Riemann sum...\n");
    integrate<<<BLOCK_NUM, THREAD_NUM>>>(deviceBlockSum, STEP_NUM, STEP_LENGTH, THREAD_NUM, BLOCK_NUM);
    sumReduce<<<1, BLOCK_NUM>>>(devicePi, deviceBlockSum, BLOCK_NUM);

    //get result to host from device
    hipMemcpy(&pi, devicePi, sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stopTime, 0);
    hipEventSynchronize(stopTime);
    float gpuTime = 0;
    hipEventElapsedTime(&gpuTime, startTime, stopTime);

    printf("Running CUDA pi approximation...\n");
    printf("PI = %.16lf with error %.16lf\nTime elapsed : %f seconds.\n\n", pi, fabs(pi - PI), gpuTime / 1000);
    assert(fabs(pi - PI) <= 0.001);

    //free memory
    hipFree(deviceBlockSum);

    hipDeviceReset();
    return 0;
}